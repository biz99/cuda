#include "hip/hip_runtime.h"
#include "common/book.h"

__global__ void add(int *a, int *b, int *c, int N){
	int tid = blockIdx.x;
	if(tid<N){
		c[tid] = a[tid] + b[tid];
	} 
}

int main(void){
	for (int N = 1000;N<=10000;N+=1000){
		// 기존: int a[N], b[N], c[N];
		int *a = (int *)malloc(N * sizeof(int));
		int *b = (int *)malloc(N * sizeof(int));
		int *c = (int *)malloc(N * sizeof(int));
		int threadsPerBlock = 256;
		int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
		int *dev_a, *dev_b, *dev_c;
		HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
		HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
		HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int)));
		for (int i = 0; i < N; i++){
			a[i] = -i;
			b[i] = i * i;
		}
		HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
		HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));
		clock_t start = clock();
		add<<<N,1>>>(dev_a, dev_b, dev_c, N);
		clock_t end = clock();
		printf("소요 시간 : %lf\n", (double)(end - start) / CLOCKS_PER_SEC);
		HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyHostToDevice));
		for(int i = 0; i<N;i++){
			//printf("%d + %d = %d \n", a[i], b[i], c[i]);
		}
		HANDLE_ERROR(hipFree(dev_a));
		HANDLE_ERROR(hipFree(dev_b));
		HANDLE_ERROR(hipFree(dev_c));
		free(a); free(b); free(c);
	}
	return 0;
}
